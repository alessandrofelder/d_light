#include "hip/hip_runtime.h"
#include <stdio.h>
#include <tiffio.h>
#include <assert.h>
#include <iostream>
#include <limits>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

typedef unsigned char GreyscaleValue; //unsigned char for 8-bit and unsigned short for 16-bit tiff

#include "flatFieldCorrect_cpu.hh"

const int gridsize = 2048;
const int blocksize = 512;

__constant__ double d_lightAverage;

__global__
void flatFieldCorrect(GreyscaleValue* d_image,GreyscaleValue* d_lightData,GreyscaleValue* d_darkData)
{
	int localIndex = blockIdx.x * blockDim.x + threadIdx.x;
	double outputVal (((double) (d_image[localIndex] - d_darkData[localIndex]))/ ((double) (d_lightData[localIndex]-d_darkData[localIndex])));
	outputVal *= d_lightAverage;
	d_image[localIndex] = (GreyscaleValue) outputVal;
}

__constant__ int d_typeMax;

__global__
void invert(GreyscaleValue* d_image)
{
	int localIndex = blockIdx.x * blockDim.x + threadIdx.x;

	d_image[localIndex] = d_typeMax - d_image[localIndex];
}

int main(int argc, const char **argv)
{
	TIFF *toCorrect	=	TIFFOpen("/home/alessandro/Documents/ImageData/070915/sloth/sloth1_00008.tif", "r");
	TIFF *corrected	=	TIFFOpen("/home/alessandro/Documents/ImageData/070915/sloth/sloth1_00008-corrected.tif", "w");

	 // initialise card
	findCudaDevice(argc, argv);

	// initialise CUDA timing

	float milli;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	//sequential reference version
	hipEventRecord(start);
	flatFieldCorrect_cpu(toCorrect, corrected);
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milli, start, stop);
	printf("\n sequential: %.1f (ms) \n", milli);

	TIFF *light = TIFFOpen("/home/alessandro/Documents/ImageData/070915/light-median-gimp.tif","r");
	TIFF *dark = TIFFOpen("/home/alessandro/Documents/ImageData/070915/dark-median-gimp.tif","r");
	TIFF *correctedGPU = TIFFOpen("/home/alessandro/Documents/ImageData/070915/sloth/sloth1_00008-correctedGPU.tif", "w");
	TIFF *invertedGPU = TIFFOpen("/home/alessandro/Documents/ImageData/070915/sloth/sloth1_00008-invertedGPU.tif", "w");
	toCorrect	=	TIFFOpen("/home/alessandro/Documents/ImageData/070915/sloth/sloth1_00008.tif", "r");

	uint32 width, height;
	uint16 bps, spp, photo, sampleFormat;
	assert(TIFFGetField(toCorrect, TIFFTAG_IMAGEWIDTH, &width));
	assert(TIFFGetField(toCorrect, TIFFTAG_IMAGELENGTH, &height));
	assert(TIFFGetField(toCorrect, TIFFTAG_BITSPERSAMPLE, &bps));
	assert(TIFFGetField(toCorrect, TIFFTAG_SAMPLESPERPIXEL, &spp));
	assert(TIFFGetField(toCorrect, TIFFTAG_PHOTOMETRIC, &photo));
	assert(TIFFGetField(toCorrect, TIFFTAG_SAMPLEFORMAT, &sampleFormat));

	assert(TIFFSetField(correctedGPU, TIFFTAG_IMAGEWIDTH, width));
	assert(TIFFSetField(correctedGPU, TIFFTAG_IMAGELENGTH, height));
	assert(TIFFSetField(correctedGPU, TIFFTAG_BITSPERSAMPLE, bps));
	assert(TIFFSetField(correctedGPU, TIFFTAG_SAMPLESPERPIXEL, spp));
	assert(TIFFSetField(correctedGPU, TIFFTAG_PHOTOMETRIC, photo));
	assert(TIFFSetField(correctedGPU, TIFFTAG_SAMPLEFORMAT, sampleFormat));

	assert(TIFFSetField(invertedGPU, TIFFTAG_IMAGEWIDTH, width));
	assert(TIFFSetField(invertedGPU, TIFFTAG_IMAGELENGTH, height));
	assert(TIFFSetField(invertedGPU, TIFFTAG_BITSPERSAMPLE, bps));
	assert(TIFFSetField(invertedGPU, TIFFTAG_SAMPLESPERPIXEL, spp));
	assert(TIFFSetField(invertedGPU, TIFFTAG_PHOTOMETRIC, photo));
	assert(TIFFSetField(invertedGPU, TIFFTAG_SAMPLEFORMAT, sampleFormat));

	int npixels = width*height;

	int linesize = TIFFScanlineSize(toCorrect);

	GreyscaleValue * h_inputData  = (GreyscaleValue *) _TIFFmalloc(linesize * width);
	GreyscaleValue * h_lightData  = (GreyscaleValue *) _TIFFmalloc(linesize * width);
	GreyscaleValue * h_darkData   = (GreyscaleValue *) _TIFFmalloc(linesize * width);
	GreyscaleValue * h_correctedData = (GreyscaleValue *) _TIFFmalloc(linesize * width);
	GreyscaleValue * h_invertedData = (GreyscaleValue *) _TIFFmalloc(linesize * width);

	double h_lightAverage = 0.0;

	for (int row = 0; row < height; row++) {
		assert(TIFFReadScanline(light, &h_lightData[row * linesize], row));
		assert(TIFFReadScanline(dark, &h_darkData[row * linesize], row));
		assert(TIFFReadScanline(toCorrect, &h_inputData[row * linesize], row));
		for (int column = 0; column < width; column++) {
			h_lightAverage += (double) h_lightData[row * linesize+column];
		}
	}

	h_lightAverage /= (npixels);

	const int dataSize = npixels*sizeof(GreyscaleValue);
	GreyscaleValue * d_data;
	GreyscaleValue * d_lightData;
	GreyscaleValue * d_darkData;

	hipEventRecord(start);

	checkCudaErrors(hipMalloc( (void**)&d_data, dataSize));
	checkCudaErrors(hipMalloc( (void**)&d_lightData, dataSize));
	checkCudaErrors(hipMalloc( (void**)&d_darkData, dataSize));

	checkCudaErrors(hipMemcpy( d_data, h_inputData, dataSize, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy( d_lightData, h_lightData, dataSize, hipMemcpyHostToDevice ));
	checkCudaErrors(hipMemcpy( d_darkData,  h_darkData,  dataSize, hipMemcpyHostToDevice ));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_lightAverage), &h_lightAverage, sizeof(double)));

	int h_typeMax = std::numeric_limits<GreyscaleValue>::max();
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_typeMax), &h_typeMax, sizeof(int)));

	dim3 dimBlock( blocksize, 1);
	dim3 dimGrid( gridsize, 1 );
	flatFieldCorrect<<<dimGrid, dimBlock>>>(d_data,d_lightData,d_darkData);
	checkCudaErrors(hipMemcpy(h_correctedData, d_data, dataSize, hipMemcpyDeviceToHost));

	checkCudaErrors(hipMemcpy( d_data, h_inputData, dataSize, hipMemcpyHostToDevice));
	invert<<<dimGrid, dimBlock>>>(d_data);
	checkCudaErrors(hipMemcpy(h_invertedData, d_data, dataSize, hipMemcpyDeviceToHost));

	checkCudaErrors(hipFree(d_data));
	checkCudaErrors(hipFree(d_lightData));
	checkCudaErrors(hipFree(d_darkData));


	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milli, start, stop);
	printf("\n gpu: %.1f (ms) \n", milli);

	for(int row=0; row<height; row++)
	{
		assert(TIFFWriteScanline(correctedGPU, &h_correctedData[row*linesize], row));
		assert(TIFFWriteScanline(invertedGPU, &h_invertedData[row*linesize], row));

	}

	hipDeviceReset();

	_TIFFfree(h_inputData);
	_TIFFfree(h_lightData);
	_TIFFfree(h_darkData);
	_TIFFfree(h_correctedData);
	_TIFFfree(h_invertedData);



	TIFFClose(light);
	TIFFClose(dark);
	TIFFClose(toCorrect);
	TIFFClose(correctedGPU);
	TIFFClose(invertedGPU);

    return 0;

}
